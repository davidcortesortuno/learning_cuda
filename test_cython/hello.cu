#include "hip/hip_runtime.h"
#include<stdio.h>
extern "C" {
    #include "hello.h"
}

__global__ void helloWorld(float f)
{
    // The thread's unique number 
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    printf("Hello block: %i threadN: %i threadIdx: %i, f=%f\n",
           blockIdx.x, threadIdx.x, idx, f);
}

int hello_f_CUDA(void) {

    helloWorld<<<2, 10>>>(1.2345f);
    hipDeviceReset();

    return  0;
}

int main(void) {

    helloWorld<<<2, 10>>>(1.2345f);
    hipDeviceReset();

    return  0;
}
