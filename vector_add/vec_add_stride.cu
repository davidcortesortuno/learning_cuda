
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void add_vec(float * a, float * b, float * res, int size)
{
    // The thread's unique number 
    int idx = threadIdx.x;
    int stride = blockDim.x;

    printf("Thread: %i Size %i Stride %i\n", idx, size, stride);

    for(int i = idx; i < size; i += stride) {
        res[i] = a[i] + b[i];
    }
    // printf("Stop\n");
}

int main(void) {

    // int N = 1<<20;
    // int * N;
    // cudaMallocManaged(&N, sizeof(int));
    int N = 20;

    float *a, *b, *result;

    // Manual mem allocation:
    // float *a, *dev_a;
    // a = (float*) malloc(sizeof(float) * N);
    // cudaMalloc((void**)&dev_a, sizeof(float) * N);

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&a, N * sizeof(float));
    hipMallocManaged(&b, N * sizeof(float));
    hipMallocManaged(&result, N * sizeof(float));

    for(int i = 0; i < N; ++i) {
        a[i] = 10.0;
        b[i] = (float) i;
    }

    add_vec<<<1, N/2>>>(a, b, result, N);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    for(int i = 0; i < N; ++i) {
        printf("%.4f\n", result[i]);
    }

    // Free memory
    hipFree(a);
    hipFree(b);
    hipFree(result);

    return  0;
}
