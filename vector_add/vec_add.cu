
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void add_vec(float * a, float * b, float * res)
{
    // The thread's unique number 
    int idx = threadIdx.x;
    printf("Thread: %i\n", idx);

    res[idx] = a[idx] + b[idx];
}

int main(void) {

    // int N = 1<<20;
    int N = 10;
    float *a, *b, *result;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&a, N * sizeof(float));
    hipMallocManaged(&b, N * sizeof(float));
    hipMallocManaged(&result, N * sizeof(float));

    for(int i = 0; i < N; ++i) {
        a[i] = 10.0;
        b[i] = (float) i;
    }

    add_vec<<<1, N>>>(a, b, result);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    for(int i = 0; i < N; ++i) {
        printf("%.4f\n", result[i]);
    }

    // Free memory
    hipFree(a);
    hipFree(b);
    hipFree(result);

    return  0;
}
